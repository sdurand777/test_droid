#include "hip/hip_runtime.h"
#include <cstdio>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>

// #include "utils.cuh"

#include <Eigen/Sparse>
#include <Eigen/SparseCore>
#include <Eigen/SparseCholesky>

typedef Eigen::SparseMatrix<double> SpMat;
typedef Eigen::Triplet<double> T;
typedef std::vector<std::vector<long>> graph_t;
typedef std::vector<torch::Tensor> tensor_list_t;



#define MIN_DEPTH 0.25

#define THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + THREADS - 1) / THREADS)


#define GPU_1D_KERNEL_LOOP(k, n) \
  for (size_t k = threadIdx.x; k<n; k += blockDim.x)


// fonction pour la somme accumulee du tableau sdata qui represente un warp soit 32 threads volatile pour ne pas optimiser sdata car plusieurs threads accedent aux data simultanement
__device__ void warpReduce(volatile float *sdata, unsigned int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid +  8];
  sdata[tid] += sdata[tid +  4];
  sdata[tid] += sdata[tid +  2];
  sdata[tid] += sdata[tid +  1];
}


// fonction pour sommer toutes les valeurs de sdata
__device__ void blockReduce(volatile float *sdata) {
  unsigned int tid = threadIdx.x;
  __syncthreads();

  // if (threadIdx.x < 256) {sdata[tid] += sdata[tid + 256]; } __syncthreads();
  if (threadIdx.x < 128) {sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (threadIdx.x <  64) {sdata[tid] += sdata[tid +  64]; } __syncthreads();

  // on arrive au niveau du warp on en profite
  if (tid < 32) warpReduce(sdata, tid);
  __syncthreads();
}



// applique une rotation a partir du quaternion q au vecteur X pour obtenir Y
__device__ void
actSO3(const float *q, const float *X, float *Y) {
  float uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}


// applique la rotation SO3 puis la translation t a X pour obtenir Y
__device__  void
actSE3(const float *t, const float *q, const float *X, float *Y) {
  actSO3(q, X, Y);
  // normalement X[3] vaut 1 scale factor pour ajouter t a Y
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}



// on applique adjoint a X pour obtenir Y attention X a 6 coords position et velocity en fait X et Y sont des jacobiens
// on connait le jaocbien X Jj et on va determiner Y Ji dans le code 
// adjoin tde SE3 matrice 6x6 avec R t^R puis 0 R
// ici on utilise la formulation avec les quaternions pour appliquer les rotations
__device__ void
adjSE3(const float *t, const float *q, const float *X, float *Y) {
    // inversion du quaternion rotation inverse
    float qinv[4] = {-q[0], -q[1], -q[2], q[3]};
    // application de la rotation inverse a X position
    actSO3(qinv, &X[0], &Y[0]);
    // application de la rotation inverse a X velocity
    actSO3(qinv, &X[3], &Y[3]);

    // compute u
    float u[3], v[3];
    u[0] = t[2]*X[1] - t[1]*X[2];
    u[1] = t[0]*X[2] - t[2]*X[0];
    u[2] = t[1]*X[0] - t[0]*X[1];

    // compute v
    actSO3(qinv, u, v);

    // compute final Y
    Y[3] += v[0];
    Y[4] += v[1];
    Y[5] += v[2];
}




// compute transfo entre deux frame S33
__device__ void 
relSE3(const float *ti, const float *qi, const float *tj, const float *qj, float *tij, float *qij) {

    // relative quaternion
    qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1],
    qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2],
    qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0],
    qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2],

    // relative translation
    actSO3(qij, ti, tij);
    tij[0] = tj[0] - tij[0];
    tij[1] = tj[1] - tij[1];
    tij[2] = tj[2] - tij[2];
}

  
// compute quaternion from phi
__device__ void
expSO3(const float *phi, float* q) {
  // SO3 exponential map
  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta_p4 = theta_sq * theta_sq;

  float theta = sqrtf(theta_sq);
  float imag, real;

  // gestion cas phi proche de 0
  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  // compute quaternion
  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;
}



// produit vectoriel a b et update b with the result
__device__ void
crossInplace(const float* a, float *b) {
  float x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}




// exponential map for SE3 conver xi en t and q xi linear algebra element to linear group SE3 elements q t on remplace R par q
__device__ void
expSE3(const float *xi, float* t, float* q) {
    // SE3 exponential map
    // get q from phi inside xi the last 3 coords of xi
    expSO3(xi + 3, q);
    // xi translation part
    float tau[3] = {xi[0], xi[1], xi[2]};
    // xi rotation part
    float phi[3] = {xi[3], xi[4], xi[5]};

    float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
    float theta = sqrtf(theta_sq);

    // compute t from tau
    t[0] = tau[0]; 
    t[1] = tau[1]; 
    t[2] = tau[2];

    // case phi  very small
    if (theta > 1e-4) {
        float a = (1 - cosf(theta)) / theta_sq;
        crossInplace(phi, tau);
        t[0] += a * tau[0];
        t[1] += a * tau[1];
        t[2] += a * tau[2];

        float b = (theta - sinf(theta)) / (theta * theta_sq);
        crossInplace(phi, tau);
        t[0] += b * tau[0];
        t[1] += b * tau[1];
        t[2] += b * tau[2];
    }
}




// build Hessian rhs vector 
__global__ void projective_transform_kernel(
        const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> target,
        const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> weight,
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
        const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
        const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
        torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> Hs,
        torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> vs,
        torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eii,
        torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eij,
        torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Cii,
        torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> bz)
{
    // recuperation indices block et thread
    // on parallelize
    // chaque block_id correspond a un edge du graph
    const int block_id = blockIdx.x;
    // les threads sont utilise au sein du block pour faire les operations en paralleles
    const int thread_id = threadIdx.x;

    // size disparite
    const int ht = disps.size(1);
    const int wd = disps.size(2);

    // indice dans le graph
    int ix = static_cast<int>(ii[block_id]);
    int jx = static_cast<int>(jj[block_id]);

    // shared data
    __shared__ float fx;
    __shared__ float fy;
    __shared__ float cx;
    __shared__ float cy;

    // relative pose for stereo frames
    __shared__ float ti[3], tj[3], tij[3];
    __shared__ float qi[4], qj[4], qij[4];

    // load intrinsics from global memory
    if (thread_id == 0) {
        fx = intrinsics[0];
        fy = intrinsics[1];
        cx = intrinsics[2];
        cy = intrinsics[3];
    }

    __syncthreads();

    // stereo frames same id
    if (ix == jx) {
        // stereo cas we fix the relative transformation between left and right
        if (thread_id == 0) {
            tij[0] =  -0.1;
            tij[1] =     0;
            tij[2] =     0;
            qij[0] =     0;
            qij[1] =     0;
            qij[2] =     0;
            qij[3] =     1;
        }
    }

    else {

        // load poses from global memory for two different lefts
        // poses 7 dimensional vector translation quaternion

        // collect translations for frame i and j
        if (thread_id < 3) {
            ti[thread_id] = poses[ix][thread_id];
            tj[thread_id] = poses[jx][thread_id];
        }

        // collect quaternion for frame i and j
        if (thread_id < 4) {
            qi[thread_id] = poses[ix][thread_id+3];
            qj[thread_id] = poses[jx][thread_id+3];
        }

        __syncthreads();
        
        // compute relatite transfo between frame i and j
        if (thread_id == 0) {
            relSE3(ti, qi, tj, qj, tij, qij);
        }
    }

    __syncthreads();

    //points  from i and j
    float Xi[4];
    float Xj[4];

    // jacobians
    // Jx tangent 12 for frame i and j tangent element in SE3 are 6 dimensional vectors rho theta

    // En fait ici on Dti Dtj Dd soit les trois jacobiens pour ce edge i j 
    // jacobien pose
    float Jx[12];
    // jacobien depth ou disparite
    float Jz;

    // frame i tangent jacobian
    float* Ji = &Jx[0];
    // frame j tangent jacobian
    float* Jj = &Jx[6];

    // hessians hij symetrique donc uniquement la moitie
    // hessiens de 12 donc uniquement pour les poses
    float hij[12*(12+1)/2];

    // vector to build the hessian problem the right handside vector - J r where here vi and vj are the pose part for frame i and j
    float vi[6], vj[6];

    int l;
    // fill with 0
    for (l=0; l<12*(12+1)/2; l++) {
        hij[l] = 0;
    }

    // fill with 0
    for (int n=0; n<6; n++) {
        vi[n] = 0;
        vj[n] = 0;
    }

    __syncthreads();

    // we loop over all pixels ht*wd
    GPU_1D_KERNEL_LOOP(k, ht*wd) {

        // pixel within ht wd image
        const int i = k / wd;
        const int j = k % wd;

        const float u = static_cast<float>(j);
        const float v = static_cast<float>(i);

        // frame i tha t we project onto frame j 

        // homogenous coordinates for each pixel
        Xi[0] = (u - cx) / fx;
        Xi[1] = (v - cy) / fy;
        Xi[2] = 1;
        Xi[3] = disps[ix][i][j];

        // transform homogenous point Xi to Xj using tij and qij soit les poses initi guess qui vont etre opitimises par le BA
        actSE3(tij, qij, Xi, Xj);

        // we het the 3D data from Xj
        const float x = Xj[0];
        const float y = Xj[1];
        const float h = Xj[3];

        // get depth from Xj and check value convert to inverse depth
        const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];

        // squered value for latter
        const float d2 = d * d;

        // confidence update based on reprojected Xj weight are from update operator convgry to weight the revised flow predicted by the convgru opertar target
        float wu = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][0][i][j];
        float wv = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][1][i][j];

        // reprojection distance between target predicted (or revised with delta to update coords) by the convgru and reprojection based on estimated disp and pose
        const float ru = target[block_id][0][i][j] - (fx * d * x + cx);
        const float rv = target[block_id][1][i][j] - (fy * d * y + cy);

        // x - coordinate

        // compute jacobian
        // h ici est un facteur echelle qui doir valoir 1
        Jj[0] = fx * (h*d);
        Jj[1] = fx * 0;
        Jj[2] = fx * (-x*h*d2);
        Jj[3] = fx * (-x*y*d2);
        Jj[4] = fx * (1 + x*x*d2);
        Jj[5] = fx * (-y*d);

        // jaocobiien par rapport a d de projection
        Jz = fx * (tij[0] * d - tij[2] * (x * d2));

        // hessian only for depth or disparity
        Cii[block_id][k] = wu * Jz * Jz;
        // bias for depth for th right hand side vector of the optimization problem
        bz[block_id][k] = wu * ru * Jz;

        if (ix == jx) wu = 0;

        // since we have Jj we compute Ji with the adjoint
        adjSE3(tij, qij, Jj, Ji);
        for (int n=0; n<6; n++) Ji[n] *= -1;

        // update hessian part for pose with Jx
        l=0;
        for (int n=0; n<12; n++) {
            for (int m=0; m<=n; m++) {
                hij[l] += wu * Jx[n] * Jx[m];
                l++;
            }
        }

        // blocks cross pose depth
        for (int n=0; n<6; n++) {
            // vector for poses right hand side
            vi[n] += wu * ru * Ji[n];
            vj[n] += wu * ru * Jj[n];

            // block of hessian cross pose depth
            Eii[block_id][n][k] = wu * Jz * Ji[n];
            Eij[block_id][n][k] = wu * Jz * Jj[n];
        }

        // y coordinate
        
        // compute jacobian from two matrices
        // pixel derivated wrt to point
        // point derivated wrt to pose
        // to get pixel derivated wrt to pose
        Jj[0] = fy * 0;
        Jj[1] = fy * (h*d);
        Jj[2] = fy * (-y*h*d2);
        Jj[3] = fy * (-1 - y*y*d2);
        Jj[4] = fy * (x*y*d2);
        Jj[5] = fy * (x*d);
        
        // jocobien suivant depth
        Jz = fy * (tij[1] * d - tij[2] * (y * d2));

        // depth hessian block
        Cii[block_id][k] += wv * Jz * Jz;
        // bias for right hand side vector
        bz[block_id][k] += wv * rv * Jz;

        if (ix == jx) wv = 0;

        // compute Ji
        adjSE3(tij, qij, Jj, Ji);
        for (int n=0; n<6; n++) Ji[n] *= -1;

        // update hessian
        l=0;
        for (int n=0; n<12; n++) {
            for (int m=0; m<=n; m++) {
                hij[l] += wv * Jx[n] * Jx[m];
                l++;
            }
        }

        // blocks cross pose and depth
        for (int n=0; n<6; n++) {
            // right hand side vector
            vi[n] += wv * rv * Ji[n];
            vj[n] += wv * rv * Jj[n];

            // cross block for the hessian
            Eii[block_id][n][k] += wv * Jz * Ji[n];
            Eij[block_id][n][k] += wv * Jz * Jj[n];
        }


    }


    // fill vs and Hs the global Hessian and global rhs vector for the optimization problem

    __syncthreads();

    __shared__ float sdata[THREADS];
    for (int n=0; n<6; n++) {
        sdata[threadIdx.x] = vi[n];
        blockReduce(sdata);
        if (threadIdx.x == 0) {
            vs[0][block_id][n] = sdata[0];
        }

        __syncthreads();

        sdata[threadIdx.x] = vj[n];
        blockReduce(sdata);
        if (threadIdx.x == 0) {
            vs[1][block_id][n] = sdata[0];
        }

    }

    l=0;
    for (int n=0; n<12; n++) {
        for (int m=0; m<=n; m++) {
            sdata[threadIdx.x] = hij[l];
            blockReduce(sdata);

            if (threadIdx.x == 0) {
                if (n<6 && m<6) {
                    // for symetry
                    Hs[0][block_id][n][m] = sdata[0];
                    Hs[0][block_id][m][n] = sdata[0];
                }
                else if (n >=6 && m<6) {
                    Hs[1][block_id][m][n-6] = sdata[0];
                    Hs[2][block_id][n-6][m] = sdata[0];
                }
                else {
                    Hs[3][block_id][n-6][m-6] = sdata[0];
                    Hs[3][block_id][m-6][n-6] = sdata[0];
                }
            }

            l++;
        }
    }
}








// method pour recuperer la distance entre les frames
__global__ void frame_distance_kernel(
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
        const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
        const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
        torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> dist,
        const float beta) 
{

    // block id
    const int block_id = blockIdx.x;
    // thread id
    const int thread_id = threadIdx.x;

    // size image
    const int ht = disps.size(1);
    const int wd = disps.size(2);

    //  id edge in the graph
    __shared__ int ix;
    __shared__ int jx;

    // intrinsics
    __shared__ float fx;
    __shared__ float fy;
    __shared__ float cx;
    __shared__ float cy;

    // transformation
    __shared__ float ti[3], tj[3], tij[3];
    __shared__ float qi[4], qj[4], qij[4];

    // load intrinsics from global memory
    if (thread_id == 0) {
        // recuperation edge
        ix = static_cast<int>(ii[block_id]);
        jx = static_cast<int>(jj[block_id]);
        // recuperation intrinsics
        fx = intrinsics[0];
        fy = intrinsics[1];
        cx = intrinsics[2];
        cy = intrinsics[3];
    }

    __syncthreads();


    //points 
    float Xi[4];
    float Xj[4];

    // temp variables
    __shared__ float accum[THREADS]; accum[thread_id] = 0;
    __shared__ float valid[THREADS]; valid[thread_id] = 0;
    __shared__ float total[THREADS]; total[thread_id] = 0;

    __syncthreads();

    // fill data
    for (int n=0; n<1; n++) {

        // recuperation des poses
        // translation
        if (thread_id < 3) {
            ti[thread_id] = poses[ix][thread_id];
            tj[thread_id] = poses[jx][thread_id];
        }

        // quaternion
        if (thread_id < 4) {
            qi[thread_id] = poses[ix][thread_id+3];
            qj[thread_id] = poses[jx][thread_id+3];
        }

        __syncthreads();

        // relative transfo between frame i and j
        relSE3(ti, qi, tj, qj, tij, qij);

        // optical flow
        float d, du, dv;

        // loop over all pixels ht * wd
        GPU_1D_KERNEL_LOOP(k, ht*wd) {
            // recuperation pixel coords
            const int i = k / wd;
            const int j = k % wd;

            // conversion to float
            const float u = static_cast<float>(j);
            const float v = static_cast<float>(i);


            // if (disps[ix][i][j] < 0.01) {
            //   continue;
            // }

            // homogenous coordinates projection 3D du pixel avec disp
            Xi[0] = (u - cx) / fx;
            Xi[1] = (v - cy) / fy;
            Xi[2] = 1;
            Xi[3] = disps[ix][i][j];

            // transform homogenous point transfo complete rotation et translation
            // reproject Xi onto frame j to obtain Xj using relative pose tij qij
            actSE3(tij, qij, Xi, Xj);

            // optical flow
            // displacement along x
            du = fx * (Xj[0] / Xj[2]) + cx - u;
            // displacement along y
            dv = fy * (Xj[1] / Xj[2]) + cy - v;
            // disp norm
            d = sqrtf(du*du + dv*dv);

            // poids entre translation et rotation
            // partie rotation ajoute
            total[threadIdx.x] += beta;

            // on a une contrainte sur la depth de la reprojection sur la frame j
            if (Xj[2] > MIN_DEPTH) {
                accum[threadIdx.x] += beta * d;
                valid[threadIdx.x] += beta;
            }

            // recuperation a nouveau de X i useless ?
            Xi[0] = (u - cx) / fx;
            Xi[1] = (v - cy) / fy;
            Xi[2] = 1;
            Xi[3] = disps[ix][i][j];

            // translation component only
            Xj[0] = Xi[0] + Xi[3] * tij[0];
            Xj[1] = Xi[1] + Xi[3] * tij[1];
            Xj[2] = Xi[2] + Xi[3] * tij[2];

            du = fx * (Xj[0] / Xj[2]) + cx - u;
            dv = fy * (Xj[1] / Xj[2]) + cy - v;
            d = sqrtf(du*du + dv*dv);

            // partie translation ajoute
            total[threadIdx.x] += (1 - beta);

            if (Xj[2] > MIN_DEPTH) {
                accum[threadIdx.x] += (1 - beta) * d;
                valid[threadIdx.x] += (1 - beta);
            }
        }

        if (threadIdx.x == 0) {
            int tmp = ix;
            ix = jx;
            jx = tmp;
        }

        __syncthreads();

    }

    // addition des distances pour chaque pixel
    __syncthreads(); blockReduce(accum);
    __syncthreads(); blockReduce(total);
    __syncthreads(); blockReduce(valid);

    __syncthreads();

    if (thread_id == 0) {
        // on check si il y a assez de pixel valid dont depth est ok sinon on fie la distance a 1000 si ok on divise accum par valid pour obtenir la moyenne
        dist[block_id] = (valid[0] / (total[0] + 1e-8) < 0.75) ? 1000.0 : accum[0] / valid[0];
    }
}







// depth filter kernel pour le visualisateur
__global__ void depth_filter_kernel(
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
        const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
        const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> inds,
        const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> thresh,
        torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> counter)
{

    // recuperation des ids
    const int block_id = blockIdx.x;
    const int neigh_id = blockIdx.y;
    // loop over index
    const int index = blockIdx.z * blockDim.x + threadIdx.x;

    // if (threadIdx.x == 0) {
    //   printf("%d %d %d %d\n", blockIdx.x, blockIdx.y, blockDim.x, threadIdx.x);
    // }

    // recuperation des dimensions de disps
    const int num = disps.size(0);
    const int ht = disps.size(1);
    const int wd = disps.size(2);

    // indices du edge
    __shared__ int ix;
    __shared__ int jx;

    // intrinsics
    __shared__ float fx;
    __shared__ float fy;
    __shared__ float cx;
    __shared__ float cy;

    // poses data for the edge frame i and frame j
    __shared__ float ti[3], tj[3], tij[3];
    __shared__ float qi[4], qj[4], qij[4];

    // intrinsics
    if (threadIdx.x == 0) {
        //  recuperation des indices
        ix = static_cast<int>(inds[block_id]);
        // on recupere jx autour de ix
        jx = (neigh_id < 3) ? ix - neigh_id - 1 : ix + neigh_id;
        // recuperation des intrinsics
        fx = intrinsics[0];
        fy = intrinsics[1];
        cx = intrinsics[2];
        cy = intrinsics[3];
    }

    __syncthreads();

    if (jx < 0 || jx >= num) 
    {
        return;
    }

    const float t = thresh[block_id];

    // load poses from global memory
    // translation
    if (threadIdx.x < 3) {
        ti[threadIdx.x] = poses[ix][threadIdx.x];
        tj[threadIdx.x] = poses[jx][threadIdx.x];
    }

    // quaternions
    if (threadIdx.x < 4) {
        qi[threadIdx.x] = poses[ix][threadIdx.x+3];
        qj[threadIdx.x] = poses[jx][threadIdx.x+3];
    }

    __syncthreads();

    if (threadIdx.x == 0) 
    {
        // transfo relative
        relSE3(ti, qi, tj, qj, tij, qij);
    }

    //points 
    float Xi[4];
    float Xj[4];

    __syncthreads();

    // loop over all pixels
    if (index < ht*wd) 
    {
        // pixel coords
        const int i = index / wd;
        const int j = index % wd;

        // pixel coords convert to float
        const float ui = static_cast<float>(j);
        const float vi = static_cast<float>(i);
        // disparity info
        const float di = disps[ix][i][j];

        // homogenous coordinates projection to 3D point from pixel
        Xi[0] = (ui - cx) / fx;
        Xi[1] = (vi - cy) / fy;
        Xi[2] = 1;
        Xi[3] = di;

        // transform homogenous point apply 3D relative transfo to project Xi onto frame j to get Xj
        actSE3(tij, qij, Xi, Xj);

        // projection of 3D Xj onto frame j to get 2d coords onto frame j
        const float uj = fx * (Xj[0] / Xj[2]) + cx;
        const float vj = fy * (Xj[1] / Xj[2]) + cy;
        const float dj = Xj[3] / Xj[2];

        // pixel cpprds on frame j
        const int u0 = static_cast<int>(floor(uj));
        const int v0 = static_cast<int>(floor(vj));

        // check if pixel coords within the image shape
        if (u0 >= 0 && v0 >= 0 && u0 < wd-1 && v0 < ht-1) 
        {
            // get decimal part
            const float wx = ceil(uj) - uj;
            const float wy = ceil(vj) - vj;

            // get depth values around pixel
            const float d00 = disps[jx][v0+0][u0+0];
            const float d01 = disps[jx][v0+0][u0+1];
            const float d10 = disps[jx][v0+1][u0+0];
            const float d11 = disps[jx][v0+1][u0+1];

            // bilinear interpolation tp get depth value
            const float dj_hat = wy*wx*d00 + wy*(1-wx)*d01 + (1-wy)*wx*d10 + (1-wy)*(1-wx)*d11;

            // filter depth based on t threshold and relatively to surrounding depth values
            const float err = abs(1.0/dj - 1.0/dj_hat);
            if       (abs(1.0/dj - 1.0/d00) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
            else if  (abs(1.0/dj - 1.0/d01) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
            else if  (abs(1.0/dj - 1.0/d10) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
            else if  (abs(1.0/dj - 1.0/d11) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
        }
    }
}



//  inverse projection kernel convert disps to points 3D
__global__ void iproj_kernel(
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
        const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
        const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
        torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> points)

{

    // get indices
    const int block_id = blockIdx.x;
    const int index = blockIdx.y * blockDim.x + threadIdx.x;

    // get disps shape info
    const int num = disps.size(0);
    const int ht = disps.size(1);
    const int wd = disps.size(2);

    // intrinsics
    __shared__ float fx;
    __shared__ float fy;
    __shared__ float cx;
    __shared__ float cy;

    // pose info
    __shared__ float t[3];
    __shared__ float q[4];

    // collect intrinsics info
    if (threadIdx.x == 0) {
        fx = intrinsics[0];
        fy = intrinsics[1];
        cx = intrinsics[2];
        cy = intrinsics[3];
    }

    __syncthreads();


    // load poses from global memory
    // fill translations
    if (threadIdx.x < 3) {
        t[threadIdx.x] = poses[block_id][threadIdx.x];
    }

    // fill quaternions
    if (threadIdx.x < 4) {
        q[threadIdx.x] = poses[block_id][threadIdx.x+3];
    }

    __syncthreads();

    //points 
    float Xi[4];
    float Xj[4];

    // loop over pixels
    if (index < ht*wd) 
    {
        // get pixel coord
        const int i = index / wd;
        const int j = index % wd;

        // convert pixel coords to float
        const float ui = static_cast<float>(j);
        const float vi = static_cast<float>(i);
        // depth value
        const float di = disps[block_id][i][j];

        // homogenous coordinates project 2D pixel of frame i into 3D point Xi
        Xi[0] = (ui - cx) / fx;
        Xi[1] = (vi - cy) / fy;
        Xi[2] = 1;
        Xi[3] = di;

        // transform homogenous point
        // apply relative transfo to Xi to project Xi onto frame j in 3D
        actSE3(t, q, Xi, Xj);

        // collect Xj
        points[block_id][i][j][0] = Xj[0] / Xj[3];
        points[block_id][i][j][1] = Xj[1] / Xj[3];
        points[block_id][i][j][2] = Xj[2] / Xj[3];

    }
}




// apply update xi to update t and q to t1 and q1
__device__ void
retrSE3(const float *xi, const float* t, const float* q, float* t1, float* q1) {
    // retraction on SE3 manifold

    float dt[3] = {0, 0, 0};
    float dq[4] = {0, 0, 0, 1};

    // convert xi to SE3 manifold dt dq
    expSE3(xi, dt, dq);

    // update q
    q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
    q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
    q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
    q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

    // update t
    actSO3(dq, t, t1);
    t1[0] += dt[0];
    t1[1] += dt[1];
    t1[2] += dt[2];
}



// recuperation de la poses updated avec dx using retrSE3
__global__ void pose_retr_kernel(
        torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dx,
        const int t0, const int t1) 
{

    for (int k=t0+threadIdx.x; k<t1; k+=blockDim.x) {
        float xi[6], q[4], q1[4], t[3], t1[3];

        t[0] = poses[k][0];
        t[1] = poses[k][1];
        t[2] = poses[k][2];

        q[0] = poses[k][3];
        q[1] = poses[k][4];
        q[2] = poses[k][5];
        q[3] = poses[k][6];

        for (int n=0; n<6; n++) {
            xi[n] = dx[k-t0][n];
        }

        // update pose t q to t1 q1
        retrSE3(xi, t, q, t1, q1);

        poses[k][0] = t1[0];
        poses[k][1] = t1[1];
        poses[k][2] = t1[2];

        poses[k][3] = q1[0];
        poses[k][4] = q1[1];
        poses[k][5] = q1[2];
        poses[k][6] = q1[3];
    }
}




// recupreation de la disps updated avec dz
__global__ void disp_retr_kernel(
        torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dz,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> inds) 
{
    const int i = inds[blockIdx.x];
    const int ht = disps.size(1);
    const int wd = disps.size(2);

    // loop over image pixels
    for (int k=threadIdx.x; k<ht*wd; k+=blockDim.x) 
    {
        // update d
        float d = disps[i][k/wd][k%wd] + dz[blockIdx.x][k];
        // update final disp with updated d
        disps[i][k/wd][k%wd] = d;
    }
}


// utils kernel on utilise ptrs et idxs pour mettre a jour inps on recupere outs
__global__ void accum_kernel(
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> inps,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ptrs,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> idxs,
        torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> outs)
{

    const int block_id = blockIdx.x;
    const int D = inps.size(2);

    const int start = ptrs[block_id];
    const int end = ptrs[block_id+1];

    for (int k=threadIdx.x; k<D; k+=blockDim.x) {
        float x = 0;
        for (int i=start; i<end; i++) {
            x += inps[idxs[i]][k];
        }
        outs[block_id][k] = x;
    }  
}


// utils accum_cuda on traite data avec ix et jx
torch::Tensor accum_cuda(torch::Tensor data, torch::Tensor ix, torch::Tensor jx) 
{
    // recuperation des indices
    torch::Tensor ix_cpu = ix.to(torch::kCPU);
    torch::Tensor jx_cpu = jx.to(torch::kCPU);
    // sort ix
    torch::Tensor inds = torch::argsort(ix_cpu);

    // init ptr vers les donnees
    long* ix_data = ix_cpu.data_ptr<long>();
    long* jx_data = jx_cpu.data_ptr<long>();
    long* kx_data = inds.data_ptr<long>();

    int count = jx.size(0);
    std::vector<int> cols;

    torch::Tensor ptrs_cpu = torch::zeros({count+1}, 
            torch::TensorOptions().dtype(torch::kInt64));

    long* ptrs_data = ptrs_cpu.data_ptr<long>();
    ptrs_data[0] = 0;

    int i = 0;
    for (int j=0; j<count; j++) {
        while (i < ix.size(0) && ix_data[kx_data[i]] <= jx_data[j]) {
            if (ix_data[kx_data[i]] == jx_data[j])
                cols.push_back(kx_data[i]);
            i++;
        }
        ptrs_data[j+1] = cols.size();
    }

    torch::Tensor idxs_cpu = torch::zeros({long(cols.size())}, 
            torch::TensorOptions().dtype(torch::kInt64));

    long* idxs_data = idxs_cpu.data_ptr<long>();

    for (int i=0; i<cols.size(); i++) {
        idxs_data[i] = cols[i];
    }

    torch::Tensor ptrs = ptrs_cpu.to(torch::kCUDA);
    torch::Tensor idxs = idxs_cpu.to(torch::kCUDA);

    torch::Tensor out = torch::zeros({jx.size(0), data.size(1)},
            torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

    // kernel cuda pour accumuler les donnees
    accum_kernel<<<count, THREADS>>>(
            data.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            ptrs.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
            idxs.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
            out.packed_accessor32<float,2,torch::RestrictPtrTraits>());

    return out;
}



// 
__global__ void EEt6x6_kernel(
        const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Q,

        const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
        torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> S)
{

    // indices
    const int ix = idx[blockIdx.x][0];
    const int jx = idx[blockIdx.x][1];
    const int kx = idx[blockIdx.x][2];

    // shape of E
    const int D = E.size(2);

    // matrice 6 * 6
    float dS[6][6];
    // gradient des poses ?
    float ei[6];
    float ej[6];

    // fill dS with 0
    for (int i=0; i<6; i++) {
        for (int j=0; j<6; j++) {
            dS[i][j] = 0;
        }
    }

    // loop over R dimension
    for (int k=threadIdx.x; k<D; k+=blockDim.x) {
        // get q data
        const float q = Q[kx][k];

        // coalesced memory read
        for (int n=0; n<6; n++) 
        {
            ei[n] = E[ix][n][k] * q;
            ej[n] = E[jx][n][k];
        }

        // block EEt
        for (int n=0; n<6; n++) {
            for (int m=0; m<6; m++) {
                dS[n][m] += ei[n] * ej[m];
            }
        }
    }

    __syncthreads();
    __shared__ float sdata[THREADS];

    // Output S
    for (int n=0; n<6; n++) {
        for (int m=0; m<6; m++) {
            sdata[threadIdx.x] = dS[n][m];

            blockReduce(sdata);

            if (threadIdx.x == 0) {
                S[blockIdx.x][n][m] = sdata[0];
            }
        }
    }
}




__global__ void Ev6x1_kernel(
        const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> E,
        const torch::PackedTensorAccessor32<float, 2,torch::RestrictPtrTraits> Q,
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w,
        const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
        torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> v)
{
    const int D = E.size(2);
    const int kx = idx[blockIdx.x][0];

    float b[6];
    for (int n=0; n<6; n++) {
        b[n] = 0.0;
    }

    for (int k=threadIdx.x; k<D; k+=blockDim.x) {
        const float q_w = Q[kx][k] * w[kx][k];

        for (int n=0; n<6; n++) {
            b[n] += q_w * E[blockIdx.x][n][k];
        }
    }

    __syncthreads();
    __shared__ float sdata[THREADS];

    for (int n=0; n<6; n++) {
        sdata[threadIdx.x] = b[n];
        blockReduce(sdata);

        if (threadIdx.x == 0) {
            v[blockIdx.x][n] += sdata[0];
        }
    }
}



__global__ void EvT6x1_kernel(
        const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
        const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> x,
        const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> idx,
        torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w)
{

    const int D = E.size(2);
    const int ix = idx[blockIdx.x];

    if (idx[blockIdx.x] <= 0 || idx[blockIdx.x] >= x.size(0))
        return;

    for (int k=threadIdx.x; k<D; k+=blockDim.x) {
        float dw = 0;
        for (int n=0; n<6; n++) {
            dw += E[blockIdx.x][n][k] * x[ix][n];
        }
        w[blockIdx.x][k] = dw;
    }
}



// class sparseblock pour build the optimization problem
class SparseBlock {
    public:

        Eigen::SparseMatrix<double> A;
        Eigen::VectorX<double> b;

        SparseBlock(int N, int M) : N(N), M(M) {
            A = Eigen::SparseMatrix<double>(N*M, N*M);
            b = Eigen::VectorXd::Zero(N*M);
        }

        SparseBlock(Eigen::SparseMatrix<double> const& A, Eigen::VectorX<double> const& b, 
                int N, int M) : A(A), b(b), N(N), M(M) {}

        // left hand side
        void update_lhs(torch::Tensor As, torch::Tensor ii, torch::Tensor jj) {

            auto As_cpu = As.to(torch::kCPU).to(torch::kFloat64);
            auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);
            auto jj_cpu = jj.to(torch::kCPU).to(torch::kInt64);

            auto As_acc = As_cpu.accessor<double,3>();
            auto ii_acc = ii_cpu.accessor<long,1>();
            auto jj_acc = jj_cpu.accessor<long,1>();

            std::vector<T> tripletList;
            for (int n=0; n<ii.size(0); n++) {
                const int i = ii_acc[n];
                const int j = jj_acc[n];

                if (i >= 0 && j >= 0) {
                    for (int k=0; k<M; k++) {
                        for (int l=0; l<M; l++) {
                            double val = As_acc[n][k][l];
                            tripletList.push_back(T(M*i + k, M*j + l, val));
                        }
                    }
                }
            }
            A.setFromTriplets(tripletList.begin(), tripletList.end());
        }

        // right hand side
        void update_rhs(torch::Tensor bs, torch::Tensor ii) {
            auto bs_cpu = bs.to(torch::kCPU).to(torch::kFloat64);
            auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);

            auto bs_acc = bs_cpu.accessor<double,2>();
            auto ii_acc = ii_cpu.accessor<long,1>();

            for (int n=0; n<ii.size(0); n++) {
                const int i = ii_acc[n];
                if (i >= 0) {
                    for (int j=0; j<M; j++) {
                        b(i*M + j) += bs_acc[n][j];
                    }
                }
            }
        }

        SparseBlock operator-(const SparseBlock& S) {
            return SparseBlock(A - S.A, b - S.b, N, M);
        }

        std::tuple<torch::Tensor, torch::Tensor> get_dense() {
            Eigen::MatrixXd Ad = Eigen::MatrixXd(A);

            torch::Tensor H = torch::from_blob(Ad.data(), {N*M, N*M}, torch::TensorOptions()
                    .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

            torch::Tensor v = torch::from_blob(b.data(), {N*M, 1}, torch::TensorOptions()
                    .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

            return std::make_tuple(H, v);

        }

        // solve optimization problem
        torch::Tensor solve(const float lm=0.0001, const float ep=0.1) {

            torch::Tensor dx;

            Eigen::SparseMatrix<double> L(A);
            L.diagonal().array() += ep + lm * L.diagonal().array();

            Eigen::SimplicialLLT<Eigen::SparseMatrix<double>> solver;
            solver.compute(L);

            if (solver.info() == Eigen::Success) {
                Eigen::VectorXd x = solver.solve(b);
                dx = torch::from_blob(x.data(), {N, M}, torch::TensorOptions()
                        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);
            }
            else {
                dx = torch::zeros({N, M}, torch::TensorOptions()
                        .device(torch::kCUDA).dtype(torch::kFloat32));
            }

            return dx;
        }

    private:
        const int N;
        const int M;

};



// schur block pour le tricks de schur
SparseBlock schur_block(torch::Tensor E,
        torch::Tensor Q,
        torch::Tensor w,
        torch::Tensor ii,
        torch::Tensor jj,
        torch::Tensor kk,
        const int t0,
        const int t1)
{

    // get indices
    torch::Tensor ii_cpu = ii.to(torch::kCPU);
    torch::Tensor jj_cpu = jj.to(torch::kCPU);
    torch::Tensor kk_cpu = kk.to(torch::kCPU);

    const int P = t1 - t0;
    const long* ii_data = ii_cpu.data_ptr<long>();
    const long* jj_data = jj_cpu.data_ptr<long>();
    const long* kk_data = kk_cpu.data_ptr<long>();

    std::vector<std::vector<long>> graph(P);
    std::vector<std::vector<long>> index(P);

    for (int n=0; n<ii_cpu.size(0); n++) {
        const int j = jj_data[n];
        const int k = kk_data[n];

        if (j >= t0 && j <= t1) {
            const int t = j - t0;
            graph[t].push_back(k);
            index[t].push_back(n);
        }
    }

    std::vector<long> ii_list, jj_list, idx, jdx;

    for (int i=0; i<P; i++) {
        for (int j=0; j<P; j++) {
            for (int k=0; k < graph[i].size(); k++) {
                for (int l=0; l < graph[j].size(); l++) {
                    if (graph[i][k] == graph[j][l]) {
                        ii_list.push_back(i);
                        jj_list.push_back(j);

                        idx.push_back(index[i][k]);
                        idx.push_back(index[j][l]);
                        idx.push_back(graph[i][k]);
                    }
                }
            }
        }
    }

    torch::Tensor ix_cuda = torch::from_blob(idx.data(), {long(idx.size())}, 
            torch::TensorOptions().dtype(torch::kInt64)).to(torch::kCUDA).view({-1, 3});

    torch::Tensor jx_cuda = torch::stack({kk_cpu}, -1)
        .to(torch::kCUDA).to(torch::kInt64);

    torch::Tensor ii2_cpu = torch::from_blob(ii_list.data(), {long(ii_list.size())}, 
            torch::TensorOptions().dtype(torch::kInt64)).view({-1});

    torch::Tensor jj2_cpu = torch::from_blob(jj_list.data(), {long(jj_list.size())}, 
            torch::TensorOptions().dtype(torch::kInt64)).view({-1});

    torch::Tensor S = torch::zeros({ix_cuda.size(0), 6, 6}, 
            torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

    torch::Tensor v = torch::zeros({jx_cuda.size(0), 6},
            torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

    EEt6x6_kernel<<<ix_cuda.size(0), THREADS>>>(
            E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
            Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            ix_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
            S.packed_accessor32<float,3,torch::RestrictPtrTraits>());

    Ev6x1_kernel<<<jx_cuda.size(0), THREADS>>>(
            E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
            Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            w.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            jx_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
            v.packed_accessor32<float,2,torch::RestrictPtrTraits>());

    // schur block
    SparseBlock A(P, 6);
    A.update_lhs(S, ii2_cpu, jj2_cpu);
    A.update_rhs(v, jj_cpu - t0);

    return A;
}








// main method for the BA
std::vector<torch::Tensor> ba_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor disps_sens,
    torch::Tensor targets,
    torch::Tensor weights,
    torch::Tensor eta,
    torch::Tensor ii,
    torch::Tensor jj,
    const int t0,
    const int t1,
    const int iterations,
    const float lm,
    const float ep,
    const bool motion_only)
{
    // // conversion to accessor
    auto targets_accessor = targets.packed_accessor32<float,4,torch::RestrictPtrTraits>();
    auto weights_accessor = weights.packed_accessor32<float,4,torch::RestrictPtrTraits>();
    auto poses_accessor =  poses.packed_accessor32<float,2,torch::RestrictPtrTraits>();
    auto disps_accessor = disps.packed_accessor32<float,3,torch::RestrictPtrTraits>();
    auto intrinsics_accessor = intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>();

    auto disps_sens_accessor = disps_sens.packed_accessor32<float,3,torch::RestrictPtrTraits>();
    auto ii_accessor = ii.packed_accessor32<long,1,torch::RestrictPtrTraits>();
    auto jj_accessor = jj.packed_accessor32<long,1,torch::RestrictPtrTraits>();

    auto eta_accessor = eta.packed_accessor32<float,3,torch::RestrictPtrTraits>();

    auto opts = poses.options();
    const int num = ii.size(0); // nombre de edges
    const int ht = disps.size(1);// shape image
    const int wd = disps.size(2); // shape image

    torch::Tensor ts = torch::arange(t0, t1).to(torch::kCUDA); // ts fpr time frame fpr BA

    std::cout << ts << std::endl;

    torch::Tensor ii_exp = torch::cat({ts, ii}, 0); 
    torch::Tensor jj_exp = torch::cat({ts, jj}, 0);

    std::cout << ii_exp << std::endl;
    std::cout << jj_exp << std::endl;



    std::tuple<torch::Tensor, torch::Tensor> kuniq = 
        torch::_unique(ii_exp, true, true);

    torch::Tensor kx = std::get<0>(kuniq);
    torch::Tensor kk_exp = std::get<1>(kuniq);

    torch::Tensor dx; // update pose
    torch::Tensor dz; // update depth

    // initialize buffers
    torch::Tensor Hs = torch::zeros({4, num, 6, 6}, opts); // hessain for pose
    torch::Tensor vs = torch::zeros({2, num, 6}, opts); // rhs vector for pose
    torch::Tensor Eii = torch::zeros({num, 6, ht*wd}, opts); // hessian for cross pose depth
    torch::Tensor Eij = torch::zeros({num, 6, ht*wd}, opts); // hessian cross block transpose
    torch::Tensor Cii = torch::zeros({num, ht*wd}, opts); // hessian block for depth
    torch::Tensor wi = torch::zeros({num, ht*wd}, opts); // confidence weight for BA

    for (int itr=0; itr<iterations; itr++) {

        projective_transform_kernel<<<num, THREADS>>>(
                targets.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
                weights.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
                poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
                disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
                intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
                ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
                jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
                Hs.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
                vs.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
                Eii.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
                Eij.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
                Cii.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
                wi.packed_accessor32<float,2,torch::RestrictPtrTraits>());


        // build the optimization problem
        SparseBlock A(t1 - t0, 6);

        // we build motion part only B for lhs et v for rhs

        // left hand side with hessian
        A.update_lhs(Hs.reshape({-1, 6, 6}), 
                torch::cat({ii, ii, jj, jj}) - t0, 
                torch::cat({ii, jj, ii, jj}) - t0);

        // right hand side with vs
        A.update_rhs(vs.reshape({-1, 6}), 
                torch::cat({ii, jj}) - t0);

        // only at the end to get non kf poses
        if (motion_only) {
            dx = A.solve(lm, ep);

            // update poses
            pose_retr_kernel<<<1, THREADS>>>(
                    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
                    dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);
        }

        else {
            // add depth residual if there are depth sensor measurements
            //const float alpha = 0.05;
            const float alpha = 0.0005;
            //printf("====== alpha 0.0005 \n");
            torch::Tensor m = (disps_sens.index({kx, "..."}) > 0).to(torch::TensorOptions().dtype(torch::kFloat32)).view({-1, ht*wd}); // mask sur les valeurs de disparites
            torch::Tensor C = accum_cuda(Cii, ii, kx) + m * alpha + (1 - m) * eta.view({-1, ht*wd}); // matrice de covariance final matrix of disp
            torch::Tensor w = accum_cuda(wi, ii, kx) - m * alpha * (disps.index({kx, "..."}) - disps_sens.index({kx, "..."})).view({-1, ht*wd}); // confidence weight
            torch::Tensor Q = 1.0 / C; // inverse matrix depth

            torch::Tensor Ei = accum_cuda(Eii.view({num, 6*ht*wd}), ii, ts).view({t1-t0, 6, ht*wd});
            torch::Tensor E = torch::cat({Ei, Eij}, 0);

            // build schur block for the depth information using E and Q
            // we have the lhs EQEt et rhs EQw
            SparseBlock S = schur_block(E, Q, w, ii_exp, jj_exp, kk_exp, t0, t1); // schur problem

            // solve for dx for the update pose
            // A - S using dedicated operator of SparseBlock provides (B - EQEt)(v - EQw) which gives the update for the pose dx 
            dx = (A - S).solve(lm, ep); // get update pose

            torch::Tensor ix = jj_exp - t0;
            torch::Tensor dw = torch::zeros({ix.size(0), ht*wd}, opts);

            EvT6x1_kernel<<<ix.size(0), THREADS>>>(
                    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
                    dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
                    ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
                    dw.packed_accessor32<float,2,torch::RestrictPtrTraits>());

            // solve for dz for the update depth
            dz = Q * (w - accum_cuda(dw, ii_exp, kx)); // update depth

            // update poses
            pose_retr_kernel<<<1, THREADS>>>(
                    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
                    dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);

            // update disparity maps
            disp_retr_kernel<<<kx.size(0), THREADS>>>(
                    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
                    dz.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
                    kx.packed_accessor32<long,1,torch::RestrictPtrTraits>());
        }

    }

    // return updates
    return {dx, dz};
}





torch::Tensor frame_distance_cuda(
        torch::Tensor poses,
        torch::Tensor disps,
        torch::Tensor intrinsics,
        torch::Tensor ii,
        torch::Tensor jj,
        const float beta)
{
    auto opts = poses.options();
    const int num = ii.size(0);

    torch::Tensor dist = torch::zeros({num}, opts);

    frame_distance_kernel<<<num, THREADS>>>(
            poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
            disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
            intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
            ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
            jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
            dist.packed_accessor32<float,1,torch::RestrictPtrTraits>(), beta);

    return dist;
}






torch::Tensor depth_filter_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ix,
    torch::Tensor thresh)
{
  const int num = ix.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor counter = torch::zeros({num, ht, wd}, disps.options());

  dim3 blocks(num, 6, NUM_BLOCKS(ht * wd));

  depth_filter_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    thresh.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    counter.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  return counter;
}




torch::Tensor iproj_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics)
{

  const int nm = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  auto opts = disps.options();
  torch::Tensor points = torch::zeros({nm, ht, wd, 3}, opts);

  dim3 blocks(nm, NUM_BLOCKS(ht * wd));

  iproj_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    points.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return points;

}





// __global__ void projmap_kernel(
//     const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
//     const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
//     const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
//     const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
//     const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
//     torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> coords,
//     torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> valid)
// {
//
//   const int block_id = blockIdx.x;
//   const int thread_id = threadIdx.x;
//
//   const int ht = disps.size(1);
//   const int wd = disps.size(2);
//
//   __shared__ int ix;
//   __shared__ int jx;
//
//   __shared__ float fx;
//   __shared__ float fy;
//   __shared__ float cx;
//   __shared__ float cy;
//
//   __shared__ float ti[3], tj[3], tij[3];
//   __shared__ float qi[4], qj[4], qij[4];
//
//   // load intrinsics from global memory
//   if (thread_id == 0) {
//     ix = static_cast<int>(ii[block_id]);
//     jx = static_cast<int>(jj[block_id]);
//     fx = intrinsics[0];
//     fy = intrinsics[1];
//     cx = intrinsics[2];
//     cy = intrinsics[3];
//   }
//
//   __syncthreads();
//
//   // load poses from global memory
//   if (thread_id < 3) {
//     ti[thread_id] = poses[ix][thread_id];
//     tj[thread_id] = poses[jx][thread_id];
//   }
//
//   if (thread_id < 4) {
//     qi[thread_id] = poses[ix][thread_id+3];
//     qj[thread_id] = poses[jx][thread_id+3];
//   }
//
//   __syncthreads();
//
//   if (thread_id == 0) {
//     relSE3(ti, qi, tj, qj, tij, qij);
//   }
//
//   //points 
//   float Xi[4];
//   float Xj[4];
//
//   __syncthreads();
//
//   GPU_1D_KERNEL_LOOP(k, ht*wd) {
//     const int i = k / wd;
//     const int j = k % wd;
//
//     const float u = static_cast<float>(j);
//     const float v = static_cast<float>(i);
//     
//     // homogenous coordinates
//     Xi[0] = (u - cx) / fx;
//     Xi[1] = (v - cy) / fy;
//     Xi[2] = 1;
//     Xi[3] = disps[ix][i][j];
//
//     // transform homogenous point
//     actSE3(tij, qij, Xi, Xj);
//
//     coords[block_id][i][j][0] = u;
//     coords[block_id][i][j][1] = v;
//
//     if (Xj[2] > 0.01) {
//       coords[block_id][i][j][0] = fx * (Xj[0] / Xj[2]) + cx;
//       coords[block_id][i][j][1] = fy * (Xj[1] / Xj[2]) + cy;
//     }
//
//     valid[block_id][i][j][0] = (Xj[2] > MIN_DEPTH) ? 1.0 : 0.0;
//
//   }
// }




// std::vector<torch::Tensor> projmap_cuda(
//     torch::Tensor poses,
//     torch::Tensor disps,
//     torch::Tensor intrinsics,
//     torch::Tensor ii,
//     torch::Tensor jj)
// {
//   auto opts = poses.options();
//   const int num = ii.size(0);
//   const int ht = disps.size(1);
//   const int wd = disps.size(2);
//
//   torch::Tensor coords = torch::zeros({num, ht, wd, 3}, opts);
//   torch::Tensor valid = torch::zeros({num, ht, wd, 1}, opts);
//
//   projmap_kernel<<<num, THREADS>>>(
//     poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
//     disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
//     intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
//     ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
//     jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
//     coords.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
//     valid.packed_accessor32<float,4,torch::RestrictPtrTraits>());
//
//   return {coords, valid};
// }



